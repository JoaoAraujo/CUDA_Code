#include "hip/hip_runtime.h"
#ifndef _KERNEL_CLUSTERS_H_
#define _KERNEL_CLUSTERS_H_

__global__ void
kernelInitCluster(cluster* in, int* numPredicates, int* subsPerBlock)
{
	in->size = *numPredicates;
	in->numSubs = 0;
	in->maxSubs = *subsPerBlock;
	in->subsPerBlock = *subsPerBlock;
}
__global__ void
kernelExpandCluster(cluster* in)
{
	in->maxSubs = in->maxSubs + in->subsPerBlock;
}
__global__ void
kernelClusterValues(cluster* c, int* result)
{
	result[0] = c->size;
	result[1] = c->maxSubs;
	result[2] = c->subsPerBlock;
}


#endif