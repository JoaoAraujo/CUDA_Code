#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <cutil_inline.h>
#include "hip/driver_types.h"
#include "hip/hip_runtime.h"
#include "clusters_struct.h"
#include <kernel_clusters.cu>


cluster* initClusters(int numberClusters, int sizePerBlock)
{
	cluster* c;
	cutilSafeCall(hipMalloc((void**) &c, sizeof(cluster)*numberClusters));//= (cluster*)malloc(sizeof(cluster)*4);
	
	int* d_sizePerBlock;
	cutilSafeCall(hipMalloc((void**) &d_sizePerBlock, sizeof(int)));
	int* d_numPreds;
	cutilSafeCall(hipMalloc((void**) &d_numPreds, sizeof(int)));
	
	cutilSafeCall(hipMemcpy(d_sizePerBlock, &sizePerBlock,sizeof(int),hipMemcpyHostToDevice));


	for(int i = 1; i <= numberClusters; i++){
		int numPreds = (int)pow((double)2,i);
		cutilSafeCall(hipMemcpy(d_numPreds, &numPreds,sizeof(int),hipMemcpyHostToDevice));
		kernelInitCluster<<<1,1>>>(&c[i-1], d_numPreds, d_sizePerBlock);
		cutilSafeCall(hipMalloc((void**) &c[i-1].subs,sizeof(int)*(numPreds*sizePerBlock+sizePerBlock)));
	}

	hipDeviceSynchronize();

	return c;
}

void expandCluster(cluster* c)
{
	int* r;
	cutilSafeCall(hipMalloc((void**) &r, sizeof(int)*3));
	int* result = (int*)malloc(sizeof(int)*3);
	kernelClusterValues<<<1,1>>>(c,r);
	cutilSafeCall(hipMemcpy(result, r, sizeof(int)*3, hipMemcpyDeviceToHost));
	int* newSubs;
	cutilSafeCall(hipMalloc((void**) &newSubs, sizeof(int)*((result[1]*result[0] + result[1])+(result[2]*result[0] + result[2]))));
	cutilSafeCall(hipMemcpy(newSubs, c->subs,((result[1]*result[0] + result[1])*sizeof(int)),hipMemcpyDeviceToDevice));
	//cutilSafeCall(hipFree(c->subs));
	cutilSafeCall(hipMalloc((void**) &c->subs, sizeof(int)*((result[1]*result[0] + result[1])+(result[2]*result[0] + result[2]))));
	cutilSafeCall(hipMemcpy(c->subs, newSubs,((result[1]*result[0] + result[1])*sizeof(int)),hipMemcpyDeviceToDevice));
	cutilSafeCall(hipFree(r));
	cutilSafeCall(hipFree(newSubs));
	free(result);
	kernelExpandCluster<<<1,1>>>(c);
	hipDeviceSynchronize();
}

void moreClusters(int currClus, int moreClus, cluster** clusters, int sizePerBlock)
{
	int* d_numPreds;
	cutilSafeCall(hipMalloc((void**) &d_numPreds, sizeof(int)));

	int* d_sizePerBlock;
	cutilSafeCall(hipMalloc((void**) &d_sizePerBlock, sizeof(int)));
	cutilSafeCall(hipMemcpy(d_sizePerBlock, &sizePerBlock,sizeof(int),hipMemcpyHostToDevice));


	for(int i = currClus; i < (moreClus+currClus); i++){
		int numPreds = (int)pow((double)2,i+1);
		cutilSafeCall(hipMemcpy(d_numPreds, &numPreds,sizeof(int),hipMemcpyHostToDevice));
		cutilSafeCall(hipMalloc((void**) &clusters[0][i], sizeof(cluster)));
		kernelInitCluster<<<1,1>>>(&clusters[0][i], d_numPreds, d_sizePerBlock);
		cutilSafeCall(hipMalloc((void**) &clusters[0][i].subs,sizeof(int)*(numPreds*sizePerBlock+sizePerBlock)));
	}

	hipDeviceSynchronize();
}