#include "hip/hip_runtime.h"
/* Example of integrating CUDA functions into an existing 
 * application / framework.
 * Device code.
 */

#ifndef _KERNELS_32_EVENTS_H_
#define _KERNELS_32_EVENTS_H_

#define ACTIVE_BIT(i, j) (i & (1 << j))
#define EDIT_BIT(i, j) (i | (1 << j))

/*extern __constant__ int3 events_d[320];//Eventos
extern __constant__ int2 sizeEvents_d[32];//Tamanho de cada Evento*/

///////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_odata  memory to process (in and out)
///////////////////////////////////////////////////////////////////////////////
__global__ void
kernelBit32EventsShared(int* tthreads, int3* predicates, int* n_predicates, float4* values, int3* anEvent, int2* sizeEvent, float4* vEvent, unsigned int* bitmap)
{
		const unsigned int tid = threadIdx.x;
		const unsigned int bid = blockIdx.x;
		const unsigned int pos = bid*blockDim.x + tid;
		const unsigned int nWarps = *tthreads/32;

		extern __shared__ float4 valuesEvent[];

		int oper;
		int atr;
		float4 val;
		float4 val2;

		if(pos == 0)
			bitmap[n_predicates[0]] = 0;


		if(tid == 0)
		{
			for(int i = 0; i < 32; i++)
			{
				int2 Event = sizeEvent[i];

				for(int k = Event.x; k < Event.y; k++)
				{
					int3 elemEvent = anEvent[k];

					for(int l = elemEvent.y; l < elemEvent.z; l+=2){
						valuesEvent[l]= vEvent[l];
						valuesEvent[l+1]= vEvent[l+1];
					}
				}
			}
		}

		__syncthreads();

		for(int i = pos; i < *n_predicates;)
		{
			int3 currPred = predicates[i];
			val = values[currPred.z];
			val2 = values[currPred.z+1];
			oper = currPred.y;
			atr = currPred.x;

			for(unsigned int j = 0; j < 32; j++)
			{
				int2 Event = sizeEvent[j];
				for(int k = Event.x; k < Event.y; k++)
				{
					int3 elemEvent = anEvent[k];

					if(elemEvent.x == atr)
					{
						for(int l = elemEvent.y; l < elemEvent.z; l+=2)
						{
							float4 currv = valuesEvent[l];
							float4 currv2 = valuesEvent[l+1];

							switch (oper)
							{
								case 0 : 
									if((val.x == currv.x && val.y == currv.y && val.z == currv.z && val.w == currv.w) && (val2.x == currv2.x && val2.y == currv2.y && val2.z == currv2.z && val2.w == currv2.w))
											bitmap[i]=EDIT_BIT(bitmap[i],j);
									break;
								case 1 :
									if(currv.x > val.x)
											bitmap[i]=EDIT_BIT(bitmap[i],j);
									break;
								case 2 :
									if(currv.x < val.x)
											bitmap[i]=EDIT_BIT(bitmap[i],j);
									break;
								case 3 :
									if((val.x != currv.x || val.y != currv.y || val.z != currv.z || val.w != currv.w) || (val2.x != currv2.x || val2.y != currv2.y || val2.z != currv2.z || val2.w != currv2.w))
											bitmap[i]=EDIT_BIT(bitmap[i],j);
									break;
								case 4 :
									if(currv.x <= val.x)
											bitmap[i]=EDIT_BIT(bitmap[i],j);
										break;
								case 5 :
									if(currv.x >= val.x)
											bitmap[i]=EDIT_BIT(bitmap[i],j);
										break;
								case 6 :
									if((val.x == currv.x && val.y == currv.y && val.z == currv.z && val.w == currv.w) && (val2.x == currv2.x && val2.y == currv2.y && val2.z == currv2.z && val2.w == currv2.w))
											bitmap[i]=EDIT_BIT(bitmap[i],j);
										break;
							}
						}
					}
				}
			}

			i += (nWarps*32);
		}
}

__global__ void
kernelBit32EventsConstant(int* tthreads, int3* predicates, int* n_predicates, float4* values, unsigned int* bitmap)
{
		const unsigned int tid = threadIdx.x;
		const unsigned int bid = blockIdx.x;
		const unsigned int pos = bid*blockDim.x + tid;
		const unsigned int nWarps = *tthreads/32;

		int oper;
		int atr;
		float4 val;
		float4 val2;

		for(int i = pos; i < *n_predicates;)
		{
			int3 currPred = predicates[i];
			val = values[currPred.z];
			val2 = values[currPred.z+1];
			oper = currPred.y;
			atr = currPred.x;

			for(unsigned int j = 0; j < 32; j++)
			{
				int2 Event = sizeEvents_d[j];
				for(int k = Event.x; k < Event.y; k++)
				{
					int3 elemEvent = events_d[k];
					if(elemEvent.x == atr)
					{
						for(int l = elemEvent.y; l < elemEvent.z; l+=2)
						{
							float4 currv = valuesEvents_d[l];
							float4 currv2 = valuesEvents_d[l+1];

							switch (oper)
							{
								case 0 : 
									if((val.x == currv.x && val.y == currv.y && val.z == currv.z && val.w == currv.w) && (val2.x == currv2.x && val2.y == currv2.y && val2.z == currv2.z && val2.w == currv2.w))
											bitmap[i]=EDIT_BIT(bitmap[i],j);
									break;
								case 1 :
									if(currv.x > val.x)
											bitmap[i]=EDIT_BIT(bitmap[i],j);
									break;
								case 2 :
									if(currv.x < val.x)
											bitmap[i]=EDIT_BIT(bitmap[i],j);
									break;
								case 3 :
									if((val.x != currv.x || val.y != currv.y || val.z != currv.z || val.w != currv.w) || (val2.x != currv2.x || val2.y != currv2.y || val2.z != currv2.z || val2.w != currv2.w))
											bitmap[i]=EDIT_BIT(bitmap[i],j);
									break;
								case 4 :
									if(currv.x <= val.x)
											bitmap[i]=EDIT_BIT(bitmap[i],j);
										break;
								case 5 :
									if(currv.x >= val.x)
											bitmap[i]=EDIT_BIT(bitmap[i],j);
										break;
								case 6 :
									if((val.x == currv.x && val.y == currv.y && val.z == currv.z && val.w == currv.w) && (val2.x == currv2.x && val2.y == currv2.y && val2.z == currv2.z && val2.w == currv2.w))
											bitmap[i]=EDIT_BIT(bitmap[i],j);
										break;
							}
						}
					}
				}
			}

			i += (nWarps*32);
		}

}

__global__ void
kernelBit32Events(int* tthreads, int3* predicates, int* n_predicates, float4* values, int3* events, int2* sizeEvent, float4* valuesEvent, unsigned int* bitmap)
{
		const unsigned int tid = threadIdx.x;
		const unsigned int bid = blockIdx.x;
		const unsigned int pos = bid*blockDim.x + tid;
		const unsigned int nWarps = *tthreads/32;

		int oper;
		int atr;
		float4 val;
		float4 val2;

		for(int i = pos; i < *n_predicates;)
		{
			int3 currPred = predicates[i];
			val = values[currPred.z];
			val2 = values[currPred.z+1];
			oper = currPred.y;
			atr = currPred.x;

			for(unsigned int j = 0; j < 32; j++)
			{
				int2 Event = sizeEvent[j];
				for(int k = Event.x; k < Event.y; k++)
				{
					int3 elemEvent = events[k];
					if(elemEvent.x == atr)
					{
						for(int l = elemEvent.y; l < elemEvent.z; l+=2)
						{
							float4 currv = valuesEvent[l];
							float4 currv2 = valuesEvent[l+1];

							switch (oper)
							{
								case 0 : 
									if((val.x == currv.x && val.y == currv.y && val.z == currv.z && val.w == currv.w) && (val2.x == currv2.x && val2.y == currv2.y && val2.z == currv2.z && val2.w == currv2.w))
											bitmap[i]=EDIT_BIT(bitmap[i],j);
									break;
								case 1 :
									if(currv.x > val.x)
											bitmap[i]=EDIT_BIT(bitmap[i],j);
									break;
								case 2 :
									if(currv.x < val.x)
											bitmap[i]=EDIT_BIT(bitmap[i],j);
									break;
								case 3 :
									if((val.x != currv.x || val.y != currv.y || val.z != currv.z || val.w != currv.w) || (val2.x != currv2.x || val2.y != currv2.y || val2.z != currv2.z || val2.w != currv2.w))
											bitmap[i]=EDIT_BIT(bitmap[i],j);
									break;
								case 4 :
									if(currv.x <= val.x)
											bitmap[i]=EDIT_BIT(bitmap[i],j);
										break;
								case 5 :
									if(currv.x >= val.x)
											bitmap[i]=EDIT_BIT(bitmap[i],j);
										break;
								case 6 :
									if((val.x == currv.x && val.y == currv.y && val.z == currv.z && val.w == currv.w) && (val2.x == currv2.x && val2.y == currv2.y && val2.z == currv2.z && val2.w == currv2.w))
											bitmap[i]=EDIT_BIT(bitmap[i],j);
										break;
							}
						}
					}
				}
			}

			i += (nWarps*32);
		}

}

__global__ void
kernelFillBit32EventShared(int* n_values, int* eventNumber, unsigned int* randomPos, unsigned int* bitmap)
{
	const unsigned int tid = threadIdx.x;
	const unsigned int bid = blockIdx.x;
	const unsigned int pos = bid*blockDim.x + tid;

		int ran;

		for(int i = pos; i < n_values[0];i++)
		{
			ran = randomPos[i];
			bitmap[ran] = EDIT_BIT(bitmap[ran], eventNumber[0]);//bitmap[pos] = 1;
		}

}

__global__ void
kernelMatch32Events(int* tthreads, unsigned int* d_bitmap, int* subs, int* c, int* size, unsigned int* result)
{
		const unsigned int tid = threadIdx.x;
		const unsigned int bid = blockIdx.x;
		const unsigned int pos = bid*blockDim.x + tid;
		const unsigned int s = (c[0]/(size[0]+1));
		const unsigned int nWarps = *tthreads/32;
		
		for(unsigned int i = pos; i < s;)
		{
			int ID = subs[i*(size[0]+1)];
			unsigned int match = d_bitmap[subs[i*(size[0]+1)+1]];

			for(unsigned int j = 2; j <= size[0] && match != 0; j++)
				match &= d_bitmap[subs[i*(size[0]+1)+j]];
		
			result[ID] = match;

			i += (nWarps*32);
		}
}

__global__ void
kernelMatch32EventsDisjunctions(int* tthreads, unsigned int* subsAccepted,unsigned int* subs, int* numSubs, int* size, unsigned int* result)
{
		const unsigned int tid = threadIdx.x;
		const unsigned int bid = blockIdx.x;
		const unsigned int pos = bid*blockDim.x + tid;
		const unsigned int nWarps = *tthreads/32;
		
		for(unsigned int i = pos; i < *numSubs;)
		{
			int ID = subs[i*(size[0]+1)];
			unsigned int match = subsAccepted[subs[i*(size[0]+1)+1]];

			for(unsigned int j = 2; j <= size[0]; j++)
				match |= subsAccepted[subs[i*(size[0]+1)+j]];
		
			result[ID] = match;

			i += (nWarps*32);
		}
}


#endif // #ifndef _CPP_INTEGRATION_KERNEL_H_