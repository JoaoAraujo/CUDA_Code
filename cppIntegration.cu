#include "hip/hip_runtime.h"
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <map>
#include <string>
#include <iostream>
#include <fstream>

#include <./include/CkCrypt2.h>
// includes, project
#include <cutil_inline.h>

#include "hip/driver_types.h"
#include "hip/hip_runtime.h"

#define NUMEVENTS 32
#define NUMATREVENTS 45

__device__ __constant__ int3 events_d[NUMEVENTS*NUMATREVENTS];//Eventos
__device__ __constant__ int2 sizeEvents_d[NUMEVENTS];//Tamanho de cada Evento
__device__ __constant__ float4 valuesEvents_d[NUMEVENTS*NUMATREVENTS*2];//Tamanho de cada Evento

// includes, kernels
#include "mem_info.h"
#include <kernels_1_Event.cu>
#include <kernels_32_Events.cu>
#include <kernels_InsRemSubs.cu>

using namespace std;

#define ACTIVE_BIT(i, j) (i & (1 << j))
#define EDIT_BIT(i, j) (i | (1 << j))

//Host
//#### AGREGADOS ####
unsigned int** clusters;
int numberClusters;

unsigned int* specialCluster;
//#### ATRIBUTOS ####
int numberAtributes;

//map<string, int> atributesRef;
int numberValues;
//map<string, int> words;

//#### SUBSCRI��ES ####
int numberSubs;
int numberSubsDisj;
map<int, int3> subscriptions;

int** newSubs;
int* numNewSubs;

map<int, int3> deletedSubs;
int* removedSubs;

//#### PREDICADOS ####
int numberPredicates;
//map<string, int> predicatesRef;
int3* predicates;
float4* valuesPrd;
//#### EVENTOS ####
int3* events;
float4* values;
int2* sizeEvents;
//#### CONFIGURA��ES ####
int totalthreads;
int num_blocks;
int num_threadsperblock;
int ntInit;
int ntFinal;
int nbInit;
int nbFinal;
int mulInit;
int mulFinal;
int bit;
int numIter;
int usingshared;
int kernelnum;
double percPerC;
int constant;

//Device
int* d_totalthreads;

__device__ unsigned int* bitmap;

int** d_clusters;
int* d_sizes;

int3* events_dev;//Eventos
int2* sizeEvents_dev;//Tamanho de cada Evento
float4* values_d;//Valores dos Eventos

int3* predicates_d;//Predicados
float4* valuesPrd_d;//Valores dos Predicados
int* numPred_d;//N�mero de Predicados

//############################ TESTS ######################################
void quickSort(unsigned int arr[], unsigned int left, unsigned int right)
{
	unsigned int i = left, j = right;

	int tmp;

	unsigned int pivot = arr[(left + right) / 2];

	/* partition */

	while (i <= j) {

		while (arr[i] < pivot)

			i++;

		while (arr[j] > pivot)

			j--;

		if (i <= j) {

			tmp = arr[i];

			arr[i] = arr[j];

			arr[j] = tmp;

			i++;

			j--;

		}

	};

	/* recursion */

	if (left < j)

		quickSort(arr, left, j);

	if (i < right)

		quickSort(arr, i, right);

}

void quickSortArrays(unsigned int arr[],unsigned int left,unsigned int right,int poww)
{
	int i = left, j = right;

	int k/* = 1*/;

	int* pivot = (int*)malloc(sizeof(int)*(poww+1));
	memcpy(pivot, &arr[((left + right) / 2)*(poww+1)], sizeof(int)*(poww+1));
	int value = pivot[1];
	/* partition */
	while (i <= j) 
	{	
		while(1)
		{
			k=1;
			while (arr[(i*(poww+1))+1] < (unsigned int)value)
				i++;

			if(arr[(i*(poww+1))+1] > (unsigned int)value)
				break;
			else if(arr[(i*(poww+1))+1] == (unsigned int)value){

				while(arr[(i*(poww+1))+k] == (unsigned int)pivot[k]){
					k++;
					if(k > poww)
						break;
				}
				if(k > poww)
					break;

				if(arr[(i*(poww+1))+k] > (unsigned int)pivot[k])
					break;
				else
					i++;
			}
		}

		while(1)
		{
			k = 1;

			while (arr[(j*(poww+1))+1] > (unsigned int)value)
				j--;

			if(arr[(j*(poww+1))+1] < (unsigned int)value )
				break;
			else if(arr[(j*(poww+1))+1] == (unsigned int)value ){

				while(arr[(j*(poww+1))+k] == pivot[k]){
					k++;

					if(k > poww)
						break;
				}
				if(k > poww)
					break;
				if(arr[(j*(poww+1))+k] < (unsigned int)pivot[k])
					break;
				else
					j--;

			}
		}

		if (i <= j) {
			int* tmp = (int*)malloc(sizeof(int)*(poww+1));
			memcpy(tmp, &arr[(i*(poww+1))],sizeof(int)*(poww+1));
			memcpy(&arr[(i*(poww+1))], &arr[(j*(poww+1))],sizeof(int)*(poww+1));
			memcpy(&arr[(j*(poww+1))], tmp,sizeof(int)*(poww+1));
			free(tmp);

			i++;
			j--;
		}
	};
	free(pivot);
	/* recursion */
	//  free(tmp);
	if (left < (unsigned int)j)

		quickSortArrays(arr, left, j, poww);

	if ((unsigned int)i < right)

		quickSortArrays(arr, i, right,poww);
}

unsigned int randomNumber(int maxV)
{
	return rand() % maxV;
}

unsigned int** randomClusters(int numSubs, int numClus, int numPred, int qs, double perc)
{
	unsigned int** r = (unsigned int**)malloc(sizeof(int)*(numClus+1));
	int i = 1;
	int init = numSubs;
	int tSubs = 0;
	int idSubs = 0;
	int init2;

	srand ( 10000 );

	r[0] = (unsigned int*)malloc(sizeof(int)*(numClus+1));

	while(i != (numClus+1))
	{
		if(i!=numClus)
			init2 = (((int)(init * perc)) >> 5) << 5;
		else
			init2 = (int)(numSubs-tSubs);

		tSubs+=init2;

		int poww = (int)pow(2.0, i);

		cout << init2 << endl;

		r[0][i] = init2*(poww+1);
		int currS = 0;

		r[i] = (unsigned int*)malloc(sizeof(int)*init2*(poww+1));

		while(currS != init2)
		{
			unsigned int* preds = (unsigned int*)malloc(sizeof(int)*(poww+1));
			preds[0] = idSubs;

			for(int j = 1; j <= poww; j++){

				int found = 1;
				unsigned int num;

				while(found == 1)
				{
					num = randomNumber(numPred);
					
					for(int k = 1; k < j; k++)
					{
						if(preds[k] == num)
						{
							found = 2;
							break;
						}
					}

					if(found == 2)
						found = 1;
					else
						found = 0;
				}
				preds[j] = num;
			}

			if(qs)
				quickSort(preds, 1, poww);

			subscriptions[idSubs].x = idSubs; 
			subscriptions[idSubs].y = i;
			subscriptions[idSubs].z = currS*(poww+1);

			memcpy(&r[i][currS*(poww+1)],preds,(poww+1)*sizeof(int));

			currS++;

			idSubs++;

			free(preds);

		}

		init-=init2;

		if(qs)
			quickSortArrays(r[i], 0, r[0][i]/(poww+1)-1, poww);

		i++;
	}
	return r;

}

unsigned int* give32Bit()
{
	unsigned int* ret = (unsigned int*)malloc(sizeof(int)*numberPredicates);
	srand ( 4294967295 );//max 32767

	for(int i = 0; i < numberPredicates; i++)
		ret[i] = (unsigned int)(rand() << 17 | rand() << 2 | rand() >> 13);

	return ret;
}

unsigned int* randomPos(double perc)
{
	srand ( 10000 );

	int size2 = static_cast<int>(numberPredicates * perc);

	//cout << size2 << endl;

	unsigned int* randomPositions = (unsigned int*)malloc(sizeof(int)*size2);

	unsigned int num;

	randomPositions[0] = randomNumber(numberPredicates);
	
		for(int i = 1; i < size2 ; i++)
		{
				int found = 1;

				while(found == 1)
				{
					num = randomNumber(numberPredicates);
					for(int j = 0; j < i; j++)
					{
						if(randomPositions[j] == num)
						{
							found = 2;
							break;
						}
					}
					if(found == 2)
						found = 1;
					else
						found = 0;
				}

				randomPositions[i] = num;
		}

	return randomPositions;

}

unsigned int** randomPercAccepted(double perc)
{
	unsigned int** ret = (unsigned int**)malloc(sizeof(unsigned int)*32);

	for(int i = 0; i < 32; i++)
		ret[i] = randomPos(perc);

	return ret;
}
//############################ PREDICATES ######################################
void initPred()
{
	srand ( 10000 );

	int pos2 = 0;
	for(int j = 0; j < numberPredicates; j++)
	{
		pos2 = j*2;
		predicates[j].x = j;
		predicates[j].y = (int)randomNumber(7);
		predicates[j].z = pos2;
	}
	int val;
	for(int i = 0; i < (numberPredicates*2);i+=2)
	{
		val = randomNumber(numberPredicates);
		valuesPrd[i].x = (float)val;
		valuesPrd[i].y = (float)val;
		valuesPrd[i].z = (float)val;
		valuesPrd[i].w = (float)val;

		valuesPrd[i+1].x = (float)val;
		valuesPrd[i+1].y = (float)val;
		valuesPrd[i+1].z = (float)val;
		valuesPrd[i+1].w = (float)val;
		/*valuesPrd[i].x = (float)i;
		valuesPrd[i].y = (float)i;
		valuesPrd[i].z = (float)i;
		valuesPrd[i].w = (float)i;

		valuesPrd[i+1].x = (float)i;
		valuesPrd[i+1].y = (float)i;
		valuesPrd[i+1].z = (float)i;
		valuesPrd[i+1].w = (float)i;*/
	}
}


//############################ EVENTS ######################################
void initEvents(int numberAtr, int numEvents)
{
	srand ( 10000 );

	int position = 0;

	for(int i = 0; i < numEvents; i++)
	{
		sizeEvents[i].x = position;

		int pos = 0;
		for(int j = 0; j < numberAtr; j++)
		{
			events[i*numberAtr + j].x = (int)randomNumber(numberPredicates); 
			events[i*numberAtr + j].y = i*(numberAtr*2)+pos;
			events[i*numberAtr + j].z = i*(numberAtr*2)+(pos+1);
			pos+=2;
		}
		position += numberAtr;
		sizeEvents[i].y = position;
	}

	for(int j = 0; j < (numEvents*numberAtr)*2; j+=2)
	{
		int ran = randomNumber(numberPredicates*2);
		values[j].x = (float)ran;
		values[j].y = (float)ran;
		values[j].z = (float)ran;
		values[j].w = (float)ran;

		values[j+1].x = (float)ran;
		values[j+1].y = (float)ran;
		values[j+1].z = (float)ran;
		values[j+1].w = (float)ran;
	}

}

//############################ BITMAP ######################################
void generateRandomBitmap(double perc)
{
		unsigned int* r = randomPos(perc);
		
		int s = (int)(numberPredicates*perc);

		int* size;
		cutilSafeCall(hipMalloc((void**) &size, sizeof(int)));
		cutilSafeCall(hipMemcpy(size, &s, sizeof(int),hipMemcpyHostToDevice));

		int* rpos;
		cutilSafeCall(hipMalloc((void**) &rpos, sizeof(int)*s));
		cutilSafeCall(hipMemcpy(rpos, r,sizeof(int)*s,hipMemcpyHostToDevice));
		
		int* d_bitsize;

		if(usingshared)
		{
			int bitsize = (int)ceil((numberPredicates+1)/32.0);
			cutilSafeCall(hipMalloc((void**) &bitmap, sizeof(unsigned int)*bitsize));
			cutilSafeCall(hipMalloc((void**) &d_bitsize, sizeof(int)));
			cutilSafeCall(hipMemcpy(d_bitsize, &bitsize,sizeof(int),hipMemcpyHostToDevice));	
			kernelFillBit1EventShared<<<1, 1>>>(size, rpos, bitmap);
		}
		else{
			cutilSafeCall(hipMalloc((void**) &bitmap, sizeof(unsigned int)*(numberPredicates+1)));
			kernelFillBit1Event<<<1, 1>>>(size, rpos, bitmap);
		}
		hipDeviceSynchronize();

		free(r);
		cutilSafeCall(hipFree(rpos));
		cutilSafeCall(hipFree(size));
}

void generateRandomBitmap32(double perc)
{
	unsigned int** r = randomPercAccepted(perc);
	
	int s = (int)(numberPredicates*perc);

	int* size;
	cutilSafeCall(hipMalloc((void**) &size, sizeof(int)));
	cutilSafeCall(hipMemcpy(size, &s, sizeof(int),hipMemcpyHostToDevice));

	unsigned int** rpos;
	cutilSafeCall(hipMalloc((void**) &rpos, sizeof(unsigned int)*s));
	cutilSafeCall(hipMemcpy(rpos, r,sizeof(unsigned int)*32,hipMemcpyHostToDevice));

	for(int i = 0; i < 32; i++)
	{
		cutilSafeCall(hipMalloc((void**) &rpos[i], sizeof(int)*s));
		cutilSafeCall(hipMemcpy(rpos[i], r[i],sizeof(int)*s,hipMemcpyHostToDevice));
	}

	cutilSafeCall(hipMalloc((void**) &bitmap, sizeof(unsigned int)*(numberPredicates+1)));

	int* eNumber;
	cutilSafeCall(hipMalloc((void**) &eNumber, sizeof(int)));

	for(int i = 0; i < 32; i++)
	{
		cutilSafeCall(hipMemcpy(eNumber, &i,sizeof(int),hipMemcpyHostToDevice));
		kernelFillBit32EventShared<<<1, 1>>>( size, eNumber, rpos[i], bitmap);
	}	
	hipDeviceSynchronize();
	
	for(int i = 0; i < 32; i++)
	{
		cutilSafeCall(hipFree(rpos[i]));
		free(r[i]);
	}

	free(r);
	cutilSafeCall(hipFree(rpos));
	cutilSafeCall(hipFree(eNumber));
	cutilSafeCall(hipFree(size));

}

void generateBitmapVar(int numEvents)
{
	ofstream f;
	f.open ("Time-Events.txt",ios::app|std::ios::out);

	unsigned int timer;
	cutilCheckError( cutCreateTimer(&timer));
	//f << "Number of predicates = " << numberPredicates << endl;
	f << "######################### EVENTS " << numEvents << " #########################" << endl;
	f << "######################### PREDICATES " << numberPredicates << " #########################" << endl;

			int numblocks= 6;
			int numthreads= 128;
		
			int totthreads = numblocks*numthreads;
			int* tt_d;

			cutilSafeCall(hipMalloc((void**) &tt_d, sizeof(int)));
			cutilSafeCall(hipMemcpy(tt_d, &totthreads,sizeof(int),hipMemcpyHostToDevice));	

			for(int i = 1; i <= 45; i++)
			{
				/*if(i%10==0)
					cout << i << endl;*/

				int numberAtr = i;
				double totaltime = 0.0;

				//cutStartTimer(timer);

				events = (int3*)malloc(sizeof(int3)*(numEvents*numberAtr));
				values = (float4*)malloc(sizeof(float4)*(numEvents*numberAtr)*2);
				sizeEvents = (int2*)malloc(sizeof(int2)*numEvents);

				predicates = (int3*)malloc(sizeof(int3)*numberPredicates);
				valuesPrd = (float4*)malloc(sizeof(float4)*(numberPredicates*2));

				initEvents(numberAtr, numEvents);
				initPred();

				cutilSafeCall(hipMalloc((void**) &predicates_d, sizeof(int3)*numberPredicates));
				cutilSafeCall(hipMalloc((void**) &valuesPrd_d, sizeof(float4)*(numberPredicates*2)));

				if(constant){
					CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(events_d), events,sizeof(int3)*(numEvents*numberAtr)));
					CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(sizeEvents_d), sizeEvents, sizeof(int2)*numEvents));
					CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(valuesEvents_d), values, sizeof(float4)*((numEvents*numberAtr)*2)));
				}
				else
				{
					cutilSafeCall(hipMalloc((void**) &events_dev, sizeof(int3)*(numEvents*numberAtr)));
					cutilSafeCall(hipMalloc((void**) &sizeEvents_dev, sizeof(int2)*numEvents));
					cutilSafeCall(hipMalloc((void**) &values_d, sizeof(float4)*((numEvents*numberAtr)*2)));

					cutilSafeCall(hipMemcpy(values_d, values,sizeof(float4)*((numEvents*numberAtr)*2),hipMemcpyHostToDevice));
					cutilSafeCall(hipMemcpy(events_dev, events,sizeof(int3)*(numEvents*numberAtr),hipMemcpyHostToDevice));
					cutilSafeCall(hipMemcpy(sizeEvents_dev, sizeEvents, sizeof(int2)*numEvents, hipMemcpyHostToDevice));
				}
				
				cutilSafeCall(hipMemcpy(predicates_d, predicates,sizeof(int3)*numberPredicates,hipMemcpyHostToDevice));		
				cutilSafeCall(hipMemcpy(valuesPrd_d, valuesPrd,sizeof(float4)*(numberPredicates*2),hipMemcpyHostToDevice));				


				//	cout << numblocks << " " << numthreads << " part = " << usingshared << " pred = " << numberPredicates << " e = " << numEvents << endl;

					for(int k = 0; k < 100; k++)
					{
						/*if(k%10 == 0)
							cout << k << endl;*/

						if(!usingshared || numEvents == 32)
							cutilSafeCall(hipMalloc((void**) &bitmap, sizeof(unsigned int)*(numberPredicates+1)));
						else
							cutilSafeCall(hipMalloc((void**) &bitmap, sizeof(unsigned int)*((int)ceil((numberPredicates+1)/32.0))));

						if(numEvents == 32){
							if(!usingshared && !constant)
							{
								cutStartTimer(timer);
								kernelBit32Events<<<numblocks, numthreads>>>(tt_d, predicates_d, numPred_d, valuesPrd_d, events_dev, sizeEvents_dev, values_d, bitmap);
								hipDeviceSynchronize();
								cutStopTimer(timer);
							}else if(usingshared && !constant){
								cutStartTimer(timer);
								kernelBit32EventsShared<<<numblocks, numthreads, sizeof(float4)*((numEvents*numberAtr)*2/* 10 KB = 16*640/1024 */)>>>(tt_d, predicates_d, numPred_d, valuesPrd_d, events_dev, sizeEvents_dev, values_d, bitmap);
								hipDeviceSynchronize();
								cutStopTimer(timer);
							}else
							{
								cutStartTimer(timer);
								kernelBit32EventsConstant<<<numblocks, numthreads>>>(tt_d, predicates_d, numPred_d, valuesPrd_d, bitmap);
								hipDeviceSynchronize();
								cutStopTimer(timer);
							}
						}
						else
						{
							if(!usingshared && !constant){
								cutStartTimer(timer);
								kernelBit1Event<<<numblocks, numthreads>>>(tt_d, predicates_d, numPred_d, valuesPrd_d, events_dev, sizeEvents_dev, values_d, bitmap);
								hipDeviceSynchronize();
								cutStopTimer(timer);
							}else if(usingshared && !constant){
								cutStartTimer(timer);
								kernelBit1EventShared<<<numblocks, numthreads, sizeof(unsigned int)*((int)ceil((numberPredicates+1)/32.0))>>>(tt_d, predicates_d, numPred_d, valuesPrd_d, events_dev, sizeEvents_dev, values_d, bitmap);
								hipDeviceSynchronize();
								cutStopTimer(timer);
							}else
							{
								cutStartTimer(timer);
								kernelBit1EventsConstant<<<numblocks, numthreads>>>(tt_d, predicates_d, numPred_d, valuesPrd_d, bitmap);
								hipDeviceSynchronize();
								cutStopTimer(timer);
							}
						}
						totaltime+=cutGetTimerValue(timer)/(double)CLOCKS_PER_SEC;
						cutResetTimer(timer);
						cutilSafeCall(hipFree(bitmap));
					}

					
			/*		hipError_t err = hipGetLastError();
					printf("Cuda error on bitmap: %s.\n", hipGetErrorString(err) );	*/
				
				free(events);
				free(values);
				free(sizeEvents);
				free(predicates);
				free(valuesPrd);
				
				if(!constant){
					cutilSafeCall(hipFree(events_dev));
					cutilSafeCall(hipFree(sizeEvents_dev));
					cutilSafeCall(hipFree(values_d));
				}
				cutilSafeCall(hipFree(predicates_d));
				cutilSafeCall(hipFree(valuesPrd_d));

				f << totaltime << endl;
		}

	cutilSafeCall(hipFree(tt_d));
}


void generateBitmap(int numblocks, int numthreads, int numEvents)
{
	//ofstream f;
	//f.open ("Time-Events.txt",ios::app|std::ios::out);

	int filename = numberPredicates+usingshared;
	ofstream f;
	char str[20];
	itoa(filename,str,10);
	f.open (str,ios::app|std::ios::out);

	int numberAtr = 10;
	unsigned int timer;
	cutilCheckError( cutCreateTimer(&timer));
	f << "Number of predicates = " << numberPredicates << endl;
	f << "######################### EVENTS " << numEvents << " #########################" << endl;


				//cutStartTimer(timer);

				events = (int3*)malloc(sizeof(int3)*(numEvents*numberAtr));
				values = (float4*)malloc(sizeof(float4)*(numEvents*numberAtr)*2);
				sizeEvents = (int2*)malloc(sizeof(int2)*numEvents);

				predicates = (int3*)malloc(sizeof(int3)*numberPredicates);
				valuesPrd = (float4*)malloc(sizeof(float4)*(numberPredicates*2));

				initEvents(numberAtr, numEvents);
				initPred();

				cutilSafeCall(hipMalloc((void**) &predicates_d, sizeof(int3)*numberPredicates));
				cutilSafeCall(hipMalloc((void**) &valuesPrd_d, sizeof(float4)*(numberPredicates*2)));
	
				if(constant){
					CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(events_d), events,sizeof(int3)*(numEvents*numberAtr)));
					CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(sizeEvents_d), sizeEvents, sizeof(int2)*numEvents));
					CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(valuesEvents_d), values, sizeof(float4)*((numEvents*numberAtr)*2)));
				}
				else
				{
					cutilSafeCall(hipMalloc((void**) &events_dev, sizeof(int3)*(numEvents*numberAtr)));
					cutilSafeCall(hipMalloc((void**) &sizeEvents_dev, sizeof(int2)*numEvents));
					cutilSafeCall(hipMalloc((void**) &values_d, sizeof(float4)*((numEvents*numberAtr)*2)));

					cutilSafeCall(hipMemcpy(values_d, values,sizeof(float4)*((numEvents*numberAtr)*2),hipMemcpyHostToDevice));
					cutilSafeCall(hipMemcpy(events_dev, events,sizeof(int3)*(numEvents*numberAtr),hipMemcpyHostToDevice));
					cutilSafeCall(hipMemcpy(sizeEvents_dev, sizeEvents, sizeof(int2)*numEvents, hipMemcpyHostToDevice));
					
				}
				
				cutilSafeCall(hipMemcpy(predicates_d, predicates,sizeof(int3)*numberPredicates,hipMemcpyHostToDevice));		
				cutilSafeCall(hipMemcpy(valuesPrd_d, valuesPrd,sizeof(float4)*(numberPredicates*2),hipMemcpyHostToDevice));				

			for(int i = 3; i < 4; i++)
			{
				f << "number blocks = " << i << endl;

				for(int j = 5; j < 6; j++)
				{
					numblocks = i;
					numthreads = pow(2.0, j);
				//	cout << numblocks << " " << numthreads << " part = " << usingshared << " pred = " << numberPredicates << " e = " << numEvents << endl;

					int totthreads = numblocks*numthreads;
					int* tt_d;
					double totaltime = 0.0;

					cutilSafeCall(hipMalloc((void**) &tt_d, sizeof(int)));
					cutilSafeCall(hipMemcpy(tt_d, &totthreads,sizeof(int),hipMemcpyHostToDevice));

					for(int k = 0; k < 1; k++)
					{
						/*if(k%10 == 0)
							cout << k << endl;*/

						if(!usingshared || numEvents == 32)
							cutilSafeCall(hipMalloc((void**) &bitmap, sizeof(unsigned int)*(numberPredicates+1)));
						else
							cutilSafeCall(hipMalloc((void**) &bitmap, sizeof(unsigned int)*((int)ceil((numberPredicates+1)/32.0))));

						if(numEvents == 32){
							if(!usingshared && !constant)
							{
								cutStartTimer(timer);
								kernelBit32Events<<<numblocks, numthreads>>>(tt_d, predicates_d, numPred_d, valuesPrd_d, events_dev, sizeEvents_dev, values_d, bitmap);
								hipDeviceSynchronize();
								cutStopTimer(timer);
							}else if(usingshared && !constant){
								cutStartTimer(timer);
								kernelBit32EventsShared<<<numblocks, numthreads, sizeof(float4)*((numEvents*numberAtr)*2/* 10 KB = 16*640/1024 */)>>>(tt_d, predicates_d, numPred_d, valuesPrd_d, events_dev, sizeEvents_dev, values_d, bitmap);
								hipDeviceSynchronize();
								cutStopTimer(timer);
							}else
							{
								cutStartTimer(timer);
								kernelBit32EventsConstant<<<numblocks, numthreads>>>(tt_d, predicates_d, numPred_d, valuesPrd_d, bitmap);
								hipDeviceSynchronize();
								cutStopTimer(timer);
							}
						}
						else
						{
							if(!usingshared && !constant){
								cutStartTimer(timer);
								kernelBit1Event<<<numblocks, numthreads>>>(tt_d, predicates_d, numPred_d, valuesPrd_d, events_dev, sizeEvents_dev, values_d, bitmap);
								hipDeviceSynchronize();
								cutStopTimer(timer);
							}else if(usingshared && !constant){
								cutStartTimer(timer);
								kernelBit1EventShared<<<numblocks, numthreads, sizeof(unsigned int)*((int)ceil((numberPredicates+1)/32.0))>>>(tt_d, predicates_d, numPred_d, valuesPrd_d, events_dev, sizeEvents_dev, values_d, bitmap);
								hipDeviceSynchronize();
								cutStopTimer(timer);
							}else
							{
								cutStartTimer(timer);
								kernelBit1EventsConstant<<<numblocks, numthreads>>>(tt_d, predicates_d, numPred_d, valuesPrd_d, bitmap);
								hipDeviceSynchronize();
								cutStopTimer(timer);
							}
						}
						totaltime+=cutGetTimerValue(timer)/(double)CLOCKS_PER_SEC;
						cutResetTimer(timer);
						//cutilSafeCall(hipFree(bitmap));
					}

					/*hipError_t err = hipGetLastError();
					printf("Cuda error on bitmap: %s.\n", hipGetErrorString(err) );	*/
					//cout << totaltime << endl;
					f << totaltime << endl;
					cutilSafeCall(hipFree(tt_d));
				}
			}
				
				free(events);
				free(values);
				free(sizeEvents);
				free(predicates);
				free(valuesPrd);
				
				if(!constant){
					cutilSafeCall(hipFree(events_dev));
					cutilSafeCall(hipFree(sizeEvents_dev));
					cutilSafeCall(hipFree(values_d));
				}
				cutilSafeCall(hipFree(predicates_d));
				cutilSafeCall(hipFree(valuesPrd_d));

			//cutilSafeCall(hipFree(bitmap));
	//cutilSafeCall(hipFree(tt_d));
}

//############################ SUBSCRIPTIONS ######################################
void initSubs(int ord)
{
	clusters = randomClusters(numberSubs, numberClusters, numberPredicates, ord, percPerC);
	int* sizes;
	sizes = (int*)malloc(sizeof(int)*(numberClusters+1));

	for(int i = 1; i <= numberClusters; i++)
		sizes[i] = (int)pow(2.0, i);//2,4,8,16,32,64,128

	cutilSafeCall(hipMalloc((void**) &d_sizes, sizeof(int)*(numberClusters+1)));
	cutilSafeCall(hipMemcpy(d_sizes, sizes,sizeof(int)*(numberClusters+1),hipMemcpyHostToDevice));

	free(sizes);

	cutilSafeCall(hipMalloc((void**) &d_clusters, sizeof(int)*(numberClusters+1)));

	cutilSafeCall(hipMalloc((void**) &d_clusters[0], sizeof(int)*(numberClusters+1)));
	cutilSafeCall(hipMemcpy(d_clusters[0], clusters[0],sizeof(int)*(numberClusters+1),hipMemcpyHostToDevice));

	for(int i = 1; i <= numberClusters; i++){
		cutilSafeCall(hipMalloc((void**) &d_clusters[i], sizeof(int)*clusters[0][i]));
		cutilSafeCall(hipMemcpy(d_clusters[i], clusters[i],sizeof(int)*clusters[0][i],hipMemcpyHostToDevice));
	}
}

void removeSubs(int begin, int numSubsToDel)
{
	srand ( 10000 );
	/*ofstrea f;
	f.open ("newSubs.txt",ios::ate);*/

	int2** subsDel;
	int2** subs = (int2**)malloc(sizeof(int2)*(numberClusters+1));

	for(int i = 1; i <= numberClusters; i++)
	{	
		subs[i] = (int2*)malloc(sizeof(int2)*(numSubsToDel+1));
		subs[i][0].x = 1;
	}
		
	int val;

	for(int i = begin; i < numSubsToDel; i++)
	{
		val = i;
		/*while(deletedSubs.count(val) != 0)
			val = rand();

		if(i%2 == 0)
			val = numberSubs - val;*/

		int2 delsub;

		delsub.x = subscriptions[val].y;
		delsub.y = subscriptions[val].z;

		subs[subscriptions[val].y][subs[subscriptions[val].y][0].x] = delsub;
		subs[subscriptions[val].y][0].x++;
		
		deletedSubs[val] = subscriptions[val];

		//numSubsDel[subscriptions[val].y]++;

		subscriptions.erase(val);
	}

	cutilSafeCall(hipMalloc((void**) &subsDel, sizeof(int2)*(numberClusters+1)));
	cutilSafeCall(hipMemcpy(subsDel, subs, sizeof(int2)*(numberClusters+1),hipMemcpyHostToDevice));

	for(int i = 1; i <= numberClusters; i++)
	{
		cutilSafeCall(hipMalloc((void**) &subsDel[i], sizeof(int2)*subs[i][0].x));
		cutilSafeCall(hipMemcpy(subsDel[i], subs[i], sizeof(int2)*subs[i][0].x,hipMemcpyHostToDevice));
	}

	for(int i = 1; i <= numberClusters; i++)
		if(subs[i][0].x > 1)
			kernelRemoveSubs<<<1,1>>>(subsDel[i], d_clusters[i], numPred_d);


	for(int i = 1; i <= numberClusters; i++){
		free(subs[i]);	
		cutilSafeCall(hipFree(subsDel[i]));
	}

	cutilSafeCall(hipFree(subsDel));
	free(subs);	

	hipDeviceSynchronize();

//	f.close();

}

void insertNewSubs(int** newSubs, int* sizeSubs, int** positions)
{
	//int newSubs[] = {numberSubs, 3426,9113, numberSubs+1, 3426, 9113};
	int**  d_newSubs;
	cutilSafeCall(hipMalloc((void**) &d_newSubs, sizeof(int)*(numberClusters+1)));

	int**  d_pos;
	cutilSafeCall(hipMalloc((void**) &d_pos, sizeof(int)*(numberClusters+1)));
	cutilSafeCall(hipMemcpy(d_pos, positions, sizeof(int)*(numberClusters+1),hipMemcpyHostToDevice));
	
	int** d_numsubs;
	cutilSafeCall(hipMalloc((void**) &d_numsubs, sizeof(int)*(numberClusters+1)));

	for(int i = 1; i <= numberClusters; i++)
	{
		cutilSafeCall(hipMalloc((void**) &d_pos[i], sizeof(int)*sizeSubs[i]));
		cutilSafeCall(hipMalloc((void**) &d_newSubs[i], sizeof(int)*(sizeSubs[i]*((int)pow(2.0,i)+1))));
		cutilSafeCall(hipMemcpy(d_newSubs[i], newSubs[i],sizeof(int)*(sizeSubs[i]*((int)pow(2.0,i)+1)),hipMemcpyHostToDevice));

		cutilSafeCall(hipMemcpy(d_pos[i], positions[i], sizeof(int)*sizeSubs[i], hipMemcpyHostToDevice));
		cutilSafeCall(hipMalloc((void**) &d_numsubs[i], sizeof(int)*(numberClusters+1)));
		cutilSafeCall(hipMemcpy(d_numsubs[i], &sizeSubs[i], sizeof(int), hipMemcpyHostToDevice));
	}

	int* size_d;
	int poww;

	for(int i = 1; i <= numberClusters; i++)
	{
		if(sizeSubs[i] > 0){
			poww = (int)pow(2.0,i);
			cutilSafeCall(hipMalloc((void**) &size_d, sizeof(int)));
			cutilSafeCall(hipMemcpy(size_d, &poww, sizeof(int), hipMemcpyHostToDevice));
			kernelInsertSubs<<<1,1>>>(d_newSubs[i], d_clusters[i], d_pos[i], d_numsubs[i], size_d);	
		}
	}

	/*hipError_t err = hipGetLastError();
	printf("Cuda error on bitmap: %s.\n", hipGetErrorString(err) );*/
	
	cutilSafeCall(hipFree(size_d));	
	hipDeviceSynchronize();
}



void obtainDisjSubs()
{
	specialCluster = (unsigned int*)malloc(sizeof(int)*(numberSubsDisj*4));

	specialCluster[0] = 0;
	specialCluster[1] = 0;
	specialCluster[2] = 1;
	specialCluster[3] = 2;

	specialCluster[4] = 1;
	specialCluster[5] = 2;
	specialCluster[6] = 5;
	specialCluster[7] = 7;

	specialCluster[8] = 2;
	specialCluster[9] = 2;
	specialCluster[10] = 11;
	specialCluster[11] = 8833;

}


void obtainMatchingDisjSubs(unsigned int* resultSubs)
{
	obtainDisjSubs();
	unsigned int* disjDevice;
	unsigned int* resultSubs_d;
	unsigned int* resultF;
	int* numSubsDisj;
	int* size_d;
	int s = 3;
	int tt = 2*32;

	cutilSafeCall(hipMalloc((void**)&resultF, sizeof(unsigned int)*numberSubsDisj));
	cutilSafeCall(hipMalloc((void**)&resultSubs_d, sizeof(unsigned int)*(numberSubs)));
	cutilSafeCall(hipMalloc((void**)&disjDevice, sizeof(unsigned int)*numberSubsDisj*4));
	cutilSafeCall(hipMalloc((void**)&d_totalthreads, sizeof(int)));
	cutilSafeCall(hipMalloc((void**)&numSubsDisj, sizeof(int)));
	cutilSafeCall(hipMalloc((void**)&size_d, sizeof(int)));

	cutilSafeCall(hipMemcpy(disjDevice, specialCluster,sizeof(unsigned int)*numberSubsDisj*4,hipMemcpyHostToDevice));
	cutilSafeCall(hipMemcpy(resultSubs_d, resultSubs,sizeof(unsigned int)*numberSubs,hipMemcpyHostToDevice));
	cutilSafeCall(hipMemcpy(d_totalthreads, &tt,sizeof(int),hipMemcpyHostToDevice));
	cutilSafeCall(hipMemcpy(numSubsDisj, &numberSubsDisj,sizeof(int),hipMemcpyHostToDevice));
	cutilSafeCall(hipMemcpy(size_d, &s,sizeof(int),hipMemcpyHostToDevice));

	kernelMatch32EventsDisjunctions<<<2,32>>>(d_totalthreads, resultSubs_d, disjDevice, numSubsDisj, size_d, resultF);

	unsigned int* r = (unsigned int*)malloc(sizeof(int)*numberSubsDisj);
	cutilSafeCall(hipMemcpy(r, resultF,sizeof(int)*numberSubsDisj,hipMemcpyDeviceToHost));

	int match = 0;

	for(int j = 0; j < 32; j++)
	{
			//fprintf(f2, "Para o evento %d foram aceites:\n", j);
			for(int i = 0; i < numberSubsDisj; i++)
			{
					unsigned int currSub = r[i];
					//cout << currSub << endl;
					if(ACTIVE_BIT(currSub, j)>0){
						cout << i << " and event = " << j << endl;
						//fprintf(f2,"%d\n",i);
						match++;
					}
			}
	}
				//cout << match << endl;
	cout << "Foram aceites " << match  << "subscricoes\n";
}
void obtainMatchingSubs(int numEvents, int perc)
{
	ofstream f;
	char str[20];
	itoa(numberPredicates,str,10);
	f.open (str,ios::app|std::ios::out);

	char str2[2];
	itoa(numEvents, str2, 10);
	FILE *f2 = fopen(str2, "a");
	
	unsigned int timer;
	cutilCheckError( cutCreateTimer(&timer));
	
	unsigned int* d_Result;

	printf("Phase 2: Start running Kernel...\n");

	cout << "######################### PREDICADOS = " << numberPredicates << " ###############################\n";
	f << "######################### NUMBER EVENTS = " << numEvents << " ###############################\n";

	if(!perc)
		f<< "---------------------------- USING BIT = RANDOM ---------------------------------" << endl;
	else
		f<< "---------------------------- USING BIT = PERC ---------------------------------" << endl;
	
	int tt;
	hipDeviceSynchronize();

	for(int mul = mulInit; mul <= mulFinal; mul +=10)
	{
			double percent = mul/100.0;
			if(perc)
			{
				f << "PERC = " << percent << endl;
				cout << "Starting with " << percent << endl;

				if(numEvents == 32)
					generateRandomBitmap32(percent);
				else
					generateRandomBitmap(percent);
			}
			else{
			//	if(nb*totalthreads <= 224)
					generateBitmap(6, 32, numEvents);
			//	else
			//		generateBitmap(nb, totalthreads, numEvents);
			}
		for(int nb = nbInit; nb < nbFinal; nb++)
		{
			num_blocks = nb;
			f << nb << " blocos" << endl;

			for(int j = ntInit; j < ntFinal; j++)
			{
				double totaltime = 0.0;		

				totalthreads = (int)pow(2.0,j);

				hipDeviceSynchronize();

				tt = totalthreads * num_blocks;
				cutilSafeCall(hipMemcpy(d_totalthreads, &tt,sizeof(int),hipMemcpyHostToDevice));

				for(int k = 0; k < numIter; k++)
				{
					cutilSafeCall(hipMalloc((void**) &d_Result, sizeof(unsigned int)*numberSubs));
					unsigned int* resultFinal = (unsigned int*)malloc(sizeof(unsigned int)*numberSubs);//copiar resultados obtidos no GPU para uma estrutura no Host

					hipDeviceSynchronize();
				
					if(numEvents == 32){
							cutStartTimer(timer);

							for(int i = 1; i <= numberClusters; i++)
								kernelMatch32Events<<<num_blocks, totalthreads>>>(d_totalthreads, bitmap, d_clusters[i], &d_clusters[0][i], &d_sizes[i], d_Result);//assincrona (devolve sem completar a execu��o)

							hipDeviceSynchronize();
							cutStopTimer(timer);
					}
					else
					{
						if(!usingshared){
							cutStartTimer(timer);

							for(int i = 1; i <= numberClusters; i++)	
								kernelMatch1Event<<<num_blocks, totalthreads>>>(d_totalthreads, bitmap, d_clusters[i], &d_clusters[0][i], &d_sizes[i], d_Result);//assincrona (devolve sem completar a execu��o)

							hipDeviceSynchronize();
							cutStopTimer(timer);
						}
						else
						{
							int* sizebit;
							cutilSafeCall(hipMalloc((void**) &sizebit, sizeof(int)));
							int sb = (int)ceil((numberPredicates+1)/32.0);
						//	cout << sb << endl;
							cutilSafeCall(hipMemcpy(sizebit, &sb,sizeof(int),hipMemcpyHostToDevice));
							cutStartTimer(timer);

							for(int i = 1; i <= numberClusters; i++)	
								kernelMatch1EventShared<<<num_blocks, totalthreads,	sizeof(int)*(int)ceil((numberPredicates+1)/32.0)>>>(d_totalthreads, bitmap, d_clusters[i], &d_clusters[0][i], &d_sizes[i], sizebit, d_Result);//assincrona (devolve sem completar a execu��o)

							hipDeviceSynchronize();
							cutStopTimer(timer);
							cutilSafeCall(hipFree(sizebit));
						}
					}

					/*hipError_t err = hipGetLastError();
					printf("Cuda error on bitmap: %s.\n", hipGetErrorString(err) );*/

					totaltime+=cutGetTimerValue(timer)/(double)CLOCKS_PER_SEC;
					cutResetTimer(timer);

					if(k%10 == 0)
						cout << nb << " " << totalthreads << " " << k << endl;

					cutilSafeCall(hipMemcpy(resultFinal, d_Result, sizeof(unsigned int)*numberSubs,hipMemcpyDeviceToHost));

					int match = 0;
					if(numEvents == 32)
					{
						for(int j = 0; j < 32; j++)
						{
							fprintf(f2, "Para o evento %d foram aceites:\n", j);
							for(int i = 0; i < numberSubs; i++)
							{
								unsigned int currSub = resultFinal[i];
								//cout << currSub << endl;
								if(ACTIVE_BIT(currSub, j)>0){
									fprintf(f2,"%d\n",i);
									//cout << currSub << endl;
									match++;
								}
							}
						}
						//cout << match << endl;
						fprintf(f2,"Foram aceites %d subscricoes\n", match);
					}
					else
					{
						for(int i = 0; i < numberSubs; i++)
						{
							int currSub = resultFinal[i];
							if(currSub > 0)
								match++;
						}
						fprintf(f2,"Foram aceites %d subscricoes\n",match);
						//cout << match << endl;
					}

					cutilSafeCall(hipFree(d_Result));
					//obtainMatchingDisjSubs(resultFinal);
					free(resultFinal);
				}

				cout << totaltime << endl;
				f << totaltime << endl;
			}
		}
		if(bit)
			cutilSafeCall(hipFree(bitmap));
	}
				
	free(str);
	free(str2);
	f.close();
	fclose(f2);
}




void firstInit(){

	//############ TAMANHO DA EXPERIENCIA ##################
	numberClusters = 7;//indicar o numero de clusters
	//numberPredicates = 1000;//numero de predicados
	numberSubs = 500000;//numero de subscricoes
	numIter = 100;//numero de itera��es
	numberSubsDisj = 3;

	//#### PERCENTAGEM DE DIVISAO DOS PREDICADOS PELAS SUBSCRICOES #####
	percPerC = 0.5;
	//############ NUM THREADS DE BASE 2 ##################
	ntInit = 8;//numero de threads iniciais 2^ntInit
	ntFinal = 9;//numero de threads finais 2^ntFinal (exclusive)
	//constant = 1;

	//############ NUM BLOCOS DE BASE ##################
	nbInit = 5;//numero de blocos iniciais
	nbFinal = 6;//numero de blocos finais (exclusive)

	//############ COM OU SEM BITMAP (USANDO OU N�O REDU��O A N�VEL DE BITS) ################
	bit = 1;//calcular o bit
	//usingshared = 1;//usar bitmap com bits ou nao

	//##### QUAL KERNEL DE MATCHING #####
	kernelnum = 1;//0 para usar o de divisao de subs pelas threads /1 -  para a versao warp /2 - versao "pior"

	cutilSafeCall(hipMalloc((void**) &d_totalthreads, sizeof(int)));
	cutilSafeCall(hipMalloc((void**) &numPred_d, sizeof(int)));
	cutilSafeCall(hipMemcpy(numPred_d, &numberPredicates,sizeof(int),hipMemcpyHostToDevice));	
	//numSubsDel = (int*)malloc(sizeof(int)*(numberClusters+1));
	/*for(int i = 0; i < (numberClusters+1); i++)
		numSubsDel[i] = 0;*/
}


void clearMemory()
{
//	free(numSubsDel);
	cutilSafeCall(hipFree(d_sizes));

	for(int i = 1; i <= numberClusters; i++)
		cutilSafeCall(hipFree(d_clusters[i]));

	cutilSafeCall(hipFree(d_clusters));

	for(int i = 0; i <= numberClusters; i++)
		free(clusters[i]);

	free(clusters);

//	cutilSafeCall(hipFree(bitmap));
	cutilSafeCall(hipFree(numPred_d));
}

int
main(int argc, char** argv)
{
	printf("Welcome to a matching publish/subscribe system! Made in CUDA!\n");

//	cout << "Running program with " << num_blocks << " blocks and " << num_threadsperblock << " threads per block = " << totalthreads << " total threads." << endl;
	
	unsigned int timer;
	cutilCheckError( cutCreateTimer(&timer));

	cutStartTimer(timer);
	usingshared = 0;
	constant = 0;
	numberPredicates = 10000;
	firstInit();
	initSubs(0);
	generateBitmap(0, 0, 1);
//	obtainMatchingSubs(1,0);
	clearMemory();
	/*cout << "PHASE 1" << endl;
	for(int i = 0; i < 2; i++){

		constant = i;
		cout << "CONSTANT: " << i << endl;
		for(int j = 1; j < 100; j*=10)
		{
			numberPredicates = 1000*j;
			cout << "PREDICATES: " << numberPredicates << endl;
			firstInit();
			initSubs(0);
			cout << "BIT" << endl;
			generateBitmap(0, 0, 1);
			cout << "OTHER BIT" << endl;
			generateBitmapVar(1);
			clearMemory();
		}
			numberPredicates = 30000;
			cout << "PREDICATES: " << numberPredicates << endl;
			firstInit();
			initSubs(0);
			cout << "BIT" << endl;
			generateBitmap(0, 0, 1);
			cout << "OTHER BIT" << endl;
			generateBitmapVar(1);
			clearMemory();

	}

	cout << "PHASE 2" << endl;
	for(int i = 0; i < 2; i++){

		constant = i;
		cout << "CONSTANT: " << i << endl;

		for(int j = 1; j < 100; j*=10)
		{
			numberPredicates = 1000*j;
			cout << "PREDICATES: " << numberPredicates << endl;
			firstInit();
			initSubs(0);
			cout << "BIT" << endl;
			generateBitmap(0, 0, 32);
			cout << "OTHER BIT" << endl;
			generateBitmapVar(32);
			clearMemory();
		}
			numberPredicates = 30000;
			cout << "PREDICATES: " << numberPredicates << endl;
			firstInit();
			initSubs(0);
			cout << "BIT" << endl;
			generateBitmap(0, 0, 32);
			cout << "OTHER BIT" << endl;
			generateBitmapVar(32);
			clearMemory();
	}
*/
	cutStopTimer(timer);
	printf("total time of execution = %f \n",cutGetTimerValue(timer)/(double)CLOCKS_PER_SEC);
	hipDeviceReset();
	//obtainMatchingSubs(32,0);
		//obtainMatchingSubs(32,1);
	//cutilExit(argc, argv);
	exit(EXIT_SUCCESS);

	return 0;
}

/*CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(c_view_planes_direction), h_view_planes_direction, sizeof(bool)*3) );
-----declara��o do array----
_constant_ bool c_view_planes_direction[3];*/