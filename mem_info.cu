#include <stdio.h>
#include <stdlib.h>
#include <malloc.h>

#include <cutil.h>

#include <hip/hip_runtime.h>

#include "hip/driver_types.h"
#include "hip/hip_runtime.h"
#include "mem_info.h"

static unsigned long inKB(unsigned long bytes)
{ return bytes/1024; }

static unsigned long inMB(unsigned long bytes)
{ return bytes/(1024*1024); }

static unsigned long inGB(unsigned long bytes)
{ return bytes/(1024*1024*1024); }


static unsigned long convert_units(unsigned long bytes, char** units)
{
	unsigned long res;
	*units = "bytes";
	if(bytes>1024)
		if(bytes>(1024*1024))
		{
			res = inMB(bytes);
			*units = "MB";
			if(bytes>(1024*1024*1024))
			{
				res = inGB(bytes);	
				*units = "GB";
			}
		}
		else
		{
			res = inKB(bytes);
			*units = "KB";
		}
	return res;
}

static void printStats(unsigned long free, unsigned long total)
{
	char* units = (char*)malloc(sizeof(char)*4);

	unsigned long converted_free = convert_units(free, &units);

	printf(" Free : %lu %s\n",converted_free, units);

	unsigned long converted_total = convert_units(total, &units);

	printf(" Total : %lu %s\n" , converted_total, units);

	printf("%g%% free, %g%% used\n\n", 100.0*free/(double)total, 100.0*(total - free)/(double)total);
}

void printGPUMemoryInfo()
{
	unsigned int free, total;
	hipMemGetInfo(&free, &total);
	printStats(free,total);
}

void printCUDAErrors(const char* id)
{
	hipError_t status = hipGetLastError();

	if( status!= hipSuccess )
	{
		printGPUMemoryInfo();
		printf("%s - %s\n",id,hipGetErrorString(status));
	//	CUT_EXIT(d_argc,d_argv);
	}
}