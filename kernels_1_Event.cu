#include "hip/hip_runtime.h"
#ifndef _KERNELS_1_EVENTS_H_
#define _KERNELS_1_EVENTS_H_

#define ACTIVE_BIT(i, j) (i & (1 << j))
#define EDIT_BIT(i, j) (i | (1 << j))

///////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_odata  memory to process (in and out)
///////////////////////////////////////////////////////////////////////////////


__global__ void
kernelBit1Event(int* tthreads, int3* predicates, int* n_predicates, float4* values, int3* anEvent, int2* sizeEvent, float4* valuesEvent, unsigned int* bitmap)
{
	const unsigned int tid = threadIdx.x;
	const unsigned int bid = blockIdx.x;
	const unsigned int pos = bid*blockDim.x + tid;
	const unsigned int nWarps = *tthreads/32;
	int oper;
	int atr;
	float4 val;
	float4 val2;

	if(pos == 0)
		bitmap[n_predicates[0]] = 0;

	for(int i = pos; i < *n_predicates; i+=(nWarps*32))
	{
		int3 currPred = predicates[i];
		val = values[currPred.z];
		val2 = values[currPred.z+1];
		oper = currPred.y;
		atr = currPred.x;

		int size = sizeEvent[0].y;

		for(int k = 0; k < size; k++)
		{
			int3 elemEvent = anEvent[k];

			if(elemEvent.x == atr)
			{
				for(int l = elemEvent.y; l < elemEvent.z; l+=2)
				{
					float4 currv = valuesEvent[l];
					float4 currv2 = valuesEvent[l+1];

					switch (oper)
					{
					case 0 : 
						if(val.x == currv.x)
							bitmap[i] = 1;
						break;
					case 1 :
						if(currv.x > val.x)
							bitmap[i] = 1;
						break;
					case 2 :
						if(currv.x < val.x)
							bitmap[i] = 1;
						break;
					case 3 :
						if(val.x != currv.x)
							bitmap[i] = 1;
						break;
					case 4 :
						if(currv.x <= val.x)
							bitmap[i] = 1;
						break;
					case 5 :
						if(currv.x >= val.x)
							bitmap[i] = 1;
						break;
					case 6 :
						if((val.x == currv.x && val.y == currv.y && val.z == currv.z && val.w == currv.w) && (val2.x == currv2.x && val2.y == currv2.y && val2.z == currv2.z && val2.w == currv2.w))
							bitmap[i] = 1;
						break;
					}
				}
			}
		}
	}
}


__global__ void
kernelFillBit1Event(int* size, int* randomPos, unsigned int* bitmap)
{
	for(int i = 0; i < size[0]; i++)
	{
		bitmap[randomPos[i]] = 1;
	}
}

__global__ void
kernelMatch1Event(int* tthreads, unsigned int* d_bitmap, int* subs, int* clusterSize, int* size, unsigned int* result)
{
	const unsigned int tid = threadIdx.x;
	const unsigned int bid = blockIdx.x;
	const unsigned int pos = bid*blockDim.x + tid;
	const unsigned int s = (clusterSize[0]/(size[0]+1));
	const unsigned int nWarps = *tthreads/32;

	for(unsigned int i = pos; i < s;)
	{
		int match = 0;
		int ID = subs[i*(size[0]+1)];

		for(unsigned int j = 1; j <= size[0]; j++)
		{
			if(d_bitmap[subs[i*(size[0]+1)+j]]==1)
				match++;
			else 
				break;
		}

		if(match == size[0]){
			result[ID] = 1;
		}

		i += (nWarps*32);
	}
}

__global__ void
kernelBit1EventShared(int* tthreads, int3* predicates, int* n_predicates, float4* values, int3* anEvent, int2* sizeEvent, float4* valuesEvent, unsigned int* bitmap)
{
	const unsigned int tid = threadIdx.x;
	const unsigned int bid = blockIdx.x;
	const unsigned int pos = bid*blockDim.x + tid;
	const unsigned int nWarps = *tthreads/32;

	int oper;
	int atr;
	float4 val;
	float4 val2;

	if(pos == 0)
		bitmap[n_predicates[0]] = 0;

	for(int i = pos; i < *n_predicates;i+=(nWarps*32))
	{
		int3 currPred = predicates[i];
		val = values[currPred.z];
		val2 = values[currPred.z+1];
		oper = currPred.y;
		atr = currPred.x;

		int size = sizeEvent[0].y;
		for(int k = 0; k < size; k++)
		{
			int3 elemEvent = anEvent[k];

			if(elemEvent.x == atr)
			{
				for(int l = elemEvent.y; l < elemEvent.z; l+=2)
				{
					float4 currv = valuesEvent[l];
					float4 currv2 = valuesEvent[l+1];

					switch (oper)
					{
					case 0 : 
						if((val.x == currv.x && val.y == currv.y && val.z == currv.z && val.w == currv.w) && (val2.x == currv2.x && val2.y == currv2.y && val2.z == currv2.z && val2.w == currv2.w))
							bitmap[i]= EDIT_BIT(bitmap[i],i);
						break;
					case 1 :
						if(currv.x > val.x)
							bitmap[i]= EDIT_BIT(bitmap[i],i);
						break;
					case 2 :
						if(currv.x < val.x)
							bitmap[i]= EDIT_BIT(bitmap[i],i);
						break;
					case 3 :
						if((val.x != currv.x || val.y != currv.y || val.z != currv.z || val.w != currv.w) || (val2.x != currv2.x || val2.y != currv2.y || val2.z != currv2.z || val2.w != currv2.w))
							bitmap[i]= EDIT_BIT(bitmap[i],i);
						break;
					case 4 :
						if(currv.x <= val.x)
							bitmap[i]= EDIT_BIT(bitmap[i],i);
						break;
					case 5 :
						if(currv.x >= val.x)
							bitmap[i]= EDIT_BIT(bitmap[i],i);
						break;
					case 6 :
						if((val.x == currv.x && val.y == currv.y && val.z == currv.z && val.w == currv.w) && (val2.x == currv2.x && val2.y == currv2.y && val2.z == currv2.z && val2.w == currv2.w))
							bitmap[i]= EDIT_BIT(bitmap[i],i);
						break;
					}
				}
			}
		}
	}
}


__global__ void
kernelFillBit1EventShared(int* size, int* randomPos, unsigned int* bitmap)
{
	int pos;

	for(int i = 0; i < size[0]; i++)
	{
		pos = randomPos[i]/32;
		bitmap[pos] = EDIT_BIT(bitmap[pos], randomPos[i]%32);//bitmap[pos] = 1;
	}
}


__global__ void
kernelMatch1EventShared(int* tthreads, unsigned int* d_bitmap, int* subs, int* c, int* size, int* bitsize, unsigned int* result)
{
	const unsigned int tid = threadIdx.x;
	const unsigned int bid = blockIdx.x;
	const unsigned int pos = bid*blockDim.x + tid;
	const unsigned int s = (c[0]/(size[0]+1));
	const unsigned int siz = size[0];
	const unsigned int nWarps = *tthreads/32;
	const unsigned int bits = *bitsize;

	extern __shared__ unsigned int b[];

	if(tid == 0)
	{
		for(int i = 0; i < bits; i++)
			b[i] = d_bitmap[i];
	}

	__syncthreads();

	for(unsigned int i = pos; i < s;)
	{
		int match = 0;
		int ID = subs[i*(siz+1)];
		int pos;

		for(unsigned int j = 1; j <= siz; j++)
		{
			pos = subs[i*(siz+1)+j];
			if((unsigned int)ACTIVE_BIT(b[pos/32], pos%32)>0)
				match++;
			else 
				break;
		}

		if(match == size[0]){
			result[ID] = 1;
		}

		i += (nWarps*32);
	}
}


__global__ void
kernelBit1EventsConstant(int* tthreads, int3* predicates, int* n_predicates, float4* values, unsigned int* bitmap)
{
		const unsigned int tid = threadIdx.x;
		const unsigned int bid = blockIdx.x;
		const unsigned int pos = bid*blockDim.x + tid;
		const unsigned int nWarps = *tthreads/32;

		int oper;
		int atr;
		float4 val;
		float4 val2;

		for(int i = pos; i < *n_predicates;)
		{
			int3 currPred = predicates[i];
			val = values[currPred.z];
			val2 = values[currPred.z+1];
			oper = currPred.y;
			atr = currPred.x;

				int size = sizeEvents_d[0].y;

				for(int k = 0; k < size; k++)
				{
					int3 elemEvent = events_d[k];

					if(elemEvent.x == atr)
					{
						for(int l = elemEvent.y; l < elemEvent.z; l+=2)
						{
							float4 currv = valuesEvents_d[l];
							float4 currv2 = valuesEvents_d[l+1];

							switch (oper)
							{
								case 0 : 
									if(val.x == currv.x)
										bitmap[i] = 1;
									break;
								case 1 :
									if(currv.x > val.x)
										bitmap[i] = 1;
									break;
								case 2 :
									if(currv.x < val.x)
										bitmap[i] = 1;
									break;
								case 3 :
									if(val.x != currv.x)
										bitmap[i] = 1;
									break;
								case 4 :
									if(currv.x <= val.x)
										bitmap[i] = 1;
									break;
								case 5 :
									if(currv.x >= val.x)
										bitmap[i] = 1;
									break;
								case 6 :
									if((val.x == currv.x && val.y == currv.y && val.z == currv.z && val.w == currv.w) && (val2.x == currv2.x && val2.y == currv2.y && val2.z == currv2.z && val2.w == currv2.w))
										bitmap[i] = 1;
									break;
							}
						}
					}
				}

			i += (nWarps*32);
		}

}

#endif // #ifndef _CPP_INTEGRATION_KERNEL_H_