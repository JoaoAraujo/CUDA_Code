
#include <hip/hip_runtime.h>
//// includes, system
//#include <stdlib.h>
//#include <stdio.h>
//#include <string.h>
//#include <math.h>
//
//#include <map>
//#include <string>
//#include <iostream>
//#include <fstream>
//// includes, project
//#include <cutil_inline.h>
//
//#include "driver_types.h"
//#include "cuda_runtime.h"
//
//using namespace std;
//
//#define ACTIVE_BIT(i, j) (i & (1 << j))
//#define EDIT_BIT(i, j) (i | (1 << j))
//
//using namespace std;
//
////Host
//unsigned int** clusters;
//int numberClusters;
//int numberSubs;
//int numberPredicates;
//int totalthreads;
//int num_blocks;
//int num_threadsperblock;
//int ntInit;
//int ntFinal;
//int nbInit;
//int nbFinal;
//int mulInit;
//int mulFinal;
//int bit;
//int numIter;
//int samePred;
//int same;
//int inverse;
//int usingshared;
//int kernelnum;
//int ran;
//int constant;
//double percPerC;
//int removedSubs;
//unsigned int* bitmap_CPU;
//
//int3* events;
//float4* values;
//int2* sizeEvents;
//int3* predicates;
//float4* valuesPrd;
//
//map<int, int3> subscriptions;
//map<int, int3> deletedSubs;
//
//unsigned int randomNumber(int maxV)
//{
//	return rand() % maxV;
//}
//
//
//unsigned int** randomClusters(int numSubs, int numClus, int numPred, int qs, double perc)
//{
//	unsigned int** r = (unsigned int**)malloc(sizeof(int)*(numClus+1));
//	int i = 1;
//	int init = numSubs;
//	int tSubs = 0;
//	int idSubs = 0;
//	int init2;
//
//	srand ( 10000 );
//
//	r[0] = (unsigned int*)malloc(sizeof(int)*(numClus+1));
//
//	while(i != (numClus+1))
//	{
//		if(i!=numClus)
//			init2 = (((int)(init * perc)) >> 5) << 5;
//		else
//			init2 = (int)(numSubs-tSubs);
//
//		tSubs+=init2;
//
//		int poww = (int)pow(2.0, i);
//
//		cout << init2 << endl;
//
//		r[0][i] = init2*(poww+1);
//		int currS = 0;
//
//		r[i] = (unsigned int*)malloc(sizeof(int)*init2*(poww+1));
//
//		while(currS != init2)
//		{
//			unsigned int* preds = (unsigned int*)malloc(sizeof(int)*(poww+1));
//			preds[0] = idSubs;
//
//			for(int j = 1; j <= poww; j++){
//
//				int found = 1;
//				unsigned int num;
//
//				while(found == 1)
//				{
//					num = randomNumber(numPred);
//					
//					for(int k = 1; k < j; k++)
//					{
//						if(preds[k] == num)
//						{
//							found = 2;
//							break;
//						}
//					}
//
//					if(found == 2)
//						found = 1;
//					else
//						found = 0;
//				}
//				preds[j] = num;
//			}
//
//			subscriptions[idSubs].x = idSubs; 
//			subscriptions[idSubs].y = i;
//			subscriptions[idSubs].z = currS*(poww+1);
//
//			memcpy(&r[i][currS*(poww+1)],preds,(poww+1)*sizeof(int));
//
//			currS++;
//
//			idSubs++;
//
//			free(preds);
//
//		}
//
//		init-=init2;
//
//		i++;
//	}
//	return r;
//
//}
//
//unsigned int* give32Bit()
//{
//	unsigned int* ret = (unsigned int*)malloc(sizeof(int)*numberPredicates);
//	srand ( 4294967295 );//max 32767
//
//	for(int i = 0; i < numberPredicates; i++)
//		ret[i] = (unsigned int)(rand() << 17 | rand() << 2 | rand() >> 13);
//
//	return ret;
//}
//
//unsigned int* randomPos(double perc)
//{
//	srand ( 10000 );
//
//	int size2 = static_cast<int>(numberPredicates * perc);
//
//	cout << size2 << endl;
//
//	unsigned int* randomPositions = (unsigned int*)malloc(sizeof(int)*size2);
//
//	unsigned int num;
//
//	randomPositions[0] = randomNumber(numberPredicates);
//	
//		for(int i = 1; i < size2 ; i++)
//		{
//				int found = 1;
//
//				while(found == 1)
//				{
//					num = randomNumber(numberPredicates);
//					for(int j = 0; j < i; j++)
//					{
//						if(randomPositions[j] == num)
//						{
//							found = 2;
//							break;
//						}
//					}
//					if(found == 2)
//						found = 1;
//					else
//						found = 0;
//				}
//
//				randomPositions[i] = num;
//		}
//
//	return randomPositions;
//
//}
//
//unsigned int** randomPercAccepted(double perc)
//{
//	unsigned int** ret = (unsigned int**)malloc(sizeof(unsigned int)*32);
//
//	for(int i = 0; i < 32; i++)
//		ret[i] = randomPos(perc);
//
//	return ret;
//}
////############################ PREDICATES ######################################
//void initPred()
//{
//	srand ( 10000 );
//
//	int pos2 = 0;
//	for(int j = 0; j < numberPredicates; j++)
//	{
//		pos2 = j*2;
//		predicates[j].x = j;
//		predicates[j].y = (int)randomNumber(7);
//		predicates[j].z = pos2;
//	}
//	for(int i = 0; i < (numberPredicates*2); i+=2)
//	{
//		valuesPrd[i].x = (float)i;
//		valuesPrd[i].y = (float)i;
//		valuesPrd[i].z = (float)i;
//		valuesPrd[i].w = (float)i;
//
//		valuesPrd[i+1].x = (float)i;
//		valuesPrd[i+1].y = (float)i;
//		valuesPrd[i+1].z = (float)i;
//		valuesPrd[i+1].w = (float)i;
//	}
//}
//
//
////############################ EVENTS ######################################
//void initEvents(int numberAtr, int numEvents)
//{
//	srand ( 10000 );
//
//	int position = 0;
//	for(int i = 0; i < numEvents; i++)
//	{
//		sizeEvents[i].x = position;
//
//		int pos = 0;
//		for(int j = 0; j < numberAtr; j++)
//		{
//			events[i*numberAtr + j].x = (int)randomNumber(numberPredicates); 
//			events[i*numberAtr + j].y = i*(numberAtr*2)+pos;
//			events[i*numberAtr + j].z = i*(numberAtr*2)+(pos+1);
//			pos+=2;
//		}
//		position += numberAtr;
//		sizeEvents[i].y = position;
//	}
//
//	for(int j = 0; j < (numEvents*numberAtr)*2; j+=2)
//	{
//		int ran = randomNumber(numberPredicates*2);
//		values[j].x = (float)ran;
//		values[j].y = (float)ran;
//		values[j].z = (float)ran;
//		values[j].w = (float)ran;
//
//		values[j+1].x = (float)ran;
//		values[j+1].y = (float)ran;
//		values[j+1].z = (float)ran;
//		values[j+1].w = (float)ran;
//	}
//
//}
//
////############################ BITMAP ######################################
//void initSubs(int ord)
//{
//	clusters = randomClusters(numberSubs, numberClusters, numberPredicates, ord, percPerC);
//}
//
//void generateRandomBitmapCPU(double perc)
//{
//		unsigned int* r = randomPos(perc);
//		
//		int s = (int)(numberPredicates*perc);
//
//		if(!usingshared){
//			bitmap_CPU = (unsigned int*)malloc(sizeof(unsigned int)*(numberPredicates+1));
//			for(int i = 0; i < numberPredicates; i++)
//				bitmap_CPU[i] = 0;
//		}
//		else
//		{
//			int bitsize = (int)ceil((numberPredicates+1)/32.0);
//			bitmap_CPU = (unsigned int*)malloc(sizeof(unsigned int)*(bitsize));
//			for(int i = 0; i < bitsize; i++)
//				bitmap_CPU[i] = 0;
//		}
//
//		int pos;
//		if(usingshared)
//		{
//			for(int i = 0; i < s; i++)
//			{
//				pos = r[i]/32;
//				bitmap_CPU[pos] = EDIT_BIT(bitmap_CPU[pos], r[i]%32);//bitmap[pos] = 1;
//			}
//		}
//		else
//		{
//			for(int i = 0; i < s; i++)
//				bitmap_CPU[r[i]] = 1;
//		}
//
//		free(r);
//}
//
//void generateRandomBitmap32CPU(double perc)
//{
//		unsigned int** r = randomPercAccepted(perc);
//		
//		int s = (int)(numberPredicates*perc);
//
//		bitmap_CPU = (unsigned int*)malloc(sizeof(unsigned int)*(numberPredicates+1));
//	
//		for(int i = 0; i < numberPredicates; i++)
//			bitmap_CPU[i] = 0;
//
//		int ran;
//
//		for(int i = 0; i < 32; i++)
//		{
//			for(int j = 0; j < s; j++)
//			{
//				ran = r[i][j];
//				bitmap_CPU[ran] = EDIT_BIT(bitmap_CPU[ran], i);//bitmap[pos] = 1;
//			}
//		}
//
//		for(int i = 0; i < 32; i++)
//			free(r[i]);
//
//		free(r);
//}
//
//
//
//
//void cpuKernelgenerateBit1()
//{
//	int oper;
//	int atr;
//	float4 val;
//	float4 val2;
//
//	bitmap_CPU[numberPredicates] = 0;
//
//	for(int i = 0; i < numberPredicates; i++)
//	{
//		int3 currPred = predicates[i];
//		val = valuesPrd[currPred.z];
//		val2 = valuesPrd[currPred.z+1];
//		oper = currPred.y;
//		atr = currPred.x;
//
//		int size = sizeEvents[0].y;
//
//		for(int k = 0; k < size; k++)
//		{
//			int3 elemEvent = events[k];
//
//			if(elemEvent.x == atr)
//			{
//				for(int l = elemEvent.y; l < elemEvent.z; l+=2)
//				{
//					float4 currv = values[l];
//					float4 currv2 = values[l+1];
//
//					switch (oper)
//					{
//					case 0 : 
//						if((val.x == currv.x && val.y == currv.y && val.z == currv.z && val.w == currv.w) && (val2.x == currv2.x && val2.y == currv2.y && val2.z == currv2.z && val2.w == currv2.w))
//							bitmap_CPU[i] = 1;
//						break;
//					case 1 :
//						if(currv.x > val.x)
//							bitmap_CPU[i] = 1;
//						break;
//					case 2 :
//						if(currv.x < val.x)
//							bitmap_CPU[i] = 1;
//						break;
//					case 3 :
//						if((val.x != currv.x || val.y != currv.y || val.z != currv.z || val.w != currv.w) || (val2.x != currv2.x || val2.y != currv2.y || val2.z != currv2.z || val2.w != currv2.w))
//							bitmap_CPU[i] = 1;
//						break;
//					case 4 :
//						if(currv.x <= val.x)
//							bitmap_CPU[i] = 1;
//						break;
//					case 5 :
//						if(currv.x >= val.x)
//							bitmap_CPU[i] = 1;
//						break;
//					case 6 :
//						if((val.x == currv.x && val.y == currv.y && val.z == currv.z && val.w == currv.w) && (val2.x == currv2.x && val2.y == currv2.y && val2.z == currv2.z && val2.w == currv2.w))
//							bitmap_CPU[i] = 1;
//						break;
//					}
//				}
//			}
//		}
//	}
//}
//void cpuKernelgenerateBit32()
//{
//		int oper;
//		int atr;
//		float4 val;
//		float4 val2;
//
//		bitmap_CPU[numberPredicates] = 0;
//
//		for(int i = 0; i < numberPredicates;i++)
//		{
//			int3 currPred = predicates[i];
//			val = valuesPrd[currPred.z];
//			val2 = valuesPrd[currPred.z+1];
//			oper = currPred.y;
//			atr = currPred.x;
//
//			for(unsigned int j = 0; j < 32; j++)
//			{
//				int2 Event = sizeEvents[j];
//
//				for(int k = Event.x; k < Event.y; k++)
//				{
//					int3 elemEvent = events[k];
//
//					if(elemEvent.x == atr)
//					{
//						for(int l = elemEvent.y; l < elemEvent.z; l+=2)
//						{
//							float4 currv = values[l];
//							float4 currv2 = values[l+1];
//
//							switch (oper)
//							{
//								case 0 : 
//									if((val.x == currv.x && val.y == currv.y && val.z == currv.z && val.w == currv.w) && (val2.x == currv2.x && val2.y == currv2.y && val2.z == currv2.z && val2.w == currv2.w))
//											bitmap_CPU[i]=EDIT_BIT(bitmap_CPU[i],j);
//									break;
//								case 1 :
//									if(currv.x > val.x)
//											bitmap_CPU[i]=EDIT_BIT(bitmap_CPU[i],j);
//									break;
//								case 2 :
//									if(currv.x < val.x)
//											bitmap_CPU[i]=EDIT_BIT(bitmap_CPU[i],j);
//									break;
//								case 3 :
//									if((val.x != currv.x || val.y != currv.y || val.z != currv.z || val.w != currv.w) || (val2.x != currv2.x || val2.y != currv2.y || val2.z != currv2.z || val2.w != currv2.w))
//										bitmap_CPU[i]=EDIT_BIT(bitmap_CPU[i],j);
//									break;
//								case 4 :
//									if(currv.x <= val.x)
//										bitmap_CPU[i]=EDIT_BIT(bitmap_CPU[i],j);
//										break;
//								case 5 :
//									if(currv.x >= val.x)
//										bitmap_CPU[i]=EDIT_BIT(bitmap_CPU[i],j);
//										break;
//								case 6 :
//									if((val.x == currv.x && val.y == currv.y && val.z == currv.z && val.w == currv.w) && (val2.x == currv2.x && val2.y == currv2.y && val2.z == currv2.z && val2.w == currv2.w))
//										bitmap_CPU[i]=EDIT_BIT(bitmap_CPU[i],j);
//										break;
//							}
//						}
//					}
//				}
//			}
//		}
//}
//void generateBitmapVar(int numEvents)
//{
//	ofstream f;
//	f.open ("Time-Events.txt",ios::app|std::ios::out);
//
//	unsigned int timer;
//	cutilCheckError( cutCreateTimer(&timer));
//	f << "Number of predicates = " << numberPredicates << endl;
//	f << "######################### EVENTS " << numEvents << " #########################" << endl;
//	//f << "######################### PREDICATES " << numberPredicates << " #########################" << endl;
//
//
//			for(int i = 1; i <= 50; i++)
//			{
//				if(i%10==0)
//					cout << i << endl;
//
//				int numberAtr = i;
//				double totaltime = 0.0;
//
//				//cutStartTimer(timer);
//
//				events = (int3*)malloc(sizeof(int3)*(numEvents*numberAtr));
//				values = (float4*)malloc(sizeof(float4)*(numEvents*numberAtr)*2);
//				sizeEvents = (int2*)malloc(sizeof(int2)*numEvents);
//
//				predicates = (int3*)malloc(sizeof(int3)*numberPredicates);
//				valuesPrd = (float4*)malloc(sizeof(float4)*(numberPredicates*2));
//
//				initEvents(numberAtr, numEvents);
//				initPred();
//
//				if(!usingshared || numEvents == 32){
//					bitmap_CPU = (unsigned int*)malloc(sizeof(unsigned int)*(numberPredicates+1));
//					for(int j = 0; j < numberPredicates; j++)
//						bitmap_CPU[j] = 0;
//				}else{
//					bitmap_CPU = (unsigned int*)malloc(sizeof(unsigned int)*((numberPredicates+1)/32.0));
//					for(int j = 0; j < (numberPredicates+1)/32.0; j++)
//						bitmap_CPU[j] = 0;
//				}
//
//				for(int k = 0; k < 100; k++)
//				{
//					if(numEvents == 32){
//							cutStartTimer(timer);
//							cpuKernelgenerateBit32();				
//							cutStopTimer(timer);
//					}else{
//						if(!usingshared){
//							cutStartTimer(timer);
//							cpuKernelgenerateBit1();
//							cutStopTimer(timer);
//						}else{
//							cutStartTimer(timer);
//							//kernelBit1EventShared<<<numblocks, numthreads, sizeof(unsigned int)*((int)ceil((numberPredicates+1)/32.0))>>>(tt_d, predicates_d, numPred_d, valuesPrd_d, events_d, sizeEvents_d, values_d, bitmap);
//							cutStopTimer(timer);
//						}
//					}
//					totaltime+=cutGetTimerValue(timer)/(double)CLOCKS_PER_SEC;
//					cutResetTimer(timer);
//				}
//
//				free(events);
//				free(values);
//				free(sizeEvents);
//				free(predicates);
//				free(valuesPrd);
//			
//				f << "Numero Atr per Event = " << i << endl;
//				f << totaltime << endl;
//
//		}
//}
//
//
//unsigned int* cpuKernels1Event()
//{
//	unsigned int* resultFinal = (unsigned int*)malloc(sizeof(unsigned int)*numberSubs);
//
//		for(int i = 1; i <= numberClusters; i++)
//		{	
//			int size = (int)pow(2.0, i);
//			int end = (clusters[0][i]/(size+1));
//
//			for(int j = 0; j < end; j++)
//			{
//				int match = 0;
//
//				int ID = clusters[i][j*(size+1)];
//
//				for(int k = 1; k <= size; k++)
//				{
//					if(bitmap_CPU[clusters[i][j*(size+1)+k]]==1)
//						match++;
//					else 
//						break;
//				}
//
//				if(match == size)
//					resultFinal[ID] = 1;	
//			}
//		}	
//
//	return resultFinal;
//}
//
//unsigned int* cpuKernels32Event()
//{
//	unsigned int* resultFinal = (unsigned int*)malloc(sizeof(unsigned int)*numberSubs);
//
//	for(int i = 1; i <= numberClusters; i++)
//	{	
//		int size = (int)pow(2.0, i);
//		int end = (clusters[0][i]/(size+1));
//
//		for(int j = 0; j < end; j++)
//		{
//				int ID = clusters[i][j*(size+1)];
//
//				unsigned int match = bitmap_CPU[clusters[i][j*(size+1)+1]];
//
//				for(int k = 2; k <= size && match != 0; k++)
//					match &= bitmap_CPU[clusters[i][j*(size+1)+k]];
//			
//				resultFinal[ID] = match;
//		}
//	}
//
//	return resultFinal;
//}
//
//unsigned int* cpuKernels1EventShared()
//{
//	unsigned int* resultFinal = (unsigned int*)malloc(sizeof(unsigned int)*numberSubs);
//
//	for(int i = 1; i <= numberClusters; i++)
//	{	
//		int size = (int)pow(2.0, i);
//		int end = (clusters[0][i]/(size+1));
//
//		for(int j = 0; j < end; j++)
//		{
//			int match = 0;
//			int ID = clusters[i][j*(size+1)];
//
//			int pos;
//
//			for(int k = 1; k <= size; k++)
//			{
//				pos = clusters[i][j*(size+1)+k];
//
//				if((unsigned int)ACTIVE_BIT(bitmap_CPU[pos/32], pos%32)>0)
//					match++;
//				else 
//					break;
//			}
//
//			if(match == size)
//				resultFinal[ID] = 1;
//		}
//	}	
//
//	return resultFinal;
//}
//
//void obtainMatchingSubsCPU(int numEvents, int perc)
//{
//	ofstream f;
//	char str[20];
//	itoa(numberPredicates,str,10);
//	f.open (str,ios::app|std::ios::out);
//
//	char str2[2];
//	itoa(numEvents, str2, 10);
//	FILE *f2 = fopen(str2, "a");
//	
//	unsigned int timer;
//	cutilCheckError( cutCreateTimer(&timer));
//
//	cout << "######################### PREDICADOS = " << numberPredicates << " ###############################\n";
//	f << "######################### NUMBER EVENTS = " << numEvents << " ###############################\n";
//
//	if(!perc)
//		f<< "---------------------------- USING BIT = RANDOM ---------------------------------" << endl;
//	else
//		f<< "---------------------------- USING BIT = PERC ---------------------------------" << endl;
//	
//	for(int mul = mulInit; mul <= mulFinal; mul +=10)
//	{
//			double percent = mul/100.0;
//
//			if(perc)
//			{
//				f << "PERC = " << percent << endl;
//				cout << "Starting with " << percent << endl;
//
//				if(numEvents == 32)
//					generateRandomBitmap32CPU(percent);
//				else
//					generateRandomBitmapCPU(percent);
//			}
//			else
//				generateBitmapVar(numEvents);
//
//				double totaltime = 0.0;		
//
//				for(int k = 0; k < numIter; k++)
//				{
//					unsigned int* result;
//
//					if(numEvents == 32){
//							cutStartTimer(timer);
//
//							result = cpuKernels32Event();
//
//							cutStopTimer(timer);
//					}
//					else
//					{
//						if(!usingshared){
//							cutStartTimer(timer);
//
//							result = cpuKernels1Event();
//
//							cutStopTimer(timer);
//						}
//						else
//						{
//							cutStartTimer(timer);
//							
//							result = cpuKernels1EventShared();
//
//							cutStopTimer(timer);
//						}
//					}
//
//					totaltime += cutGetTimerValue(timer)/(double)CLOCKS_PER_SEC;
//					cutResetTimer(timer);
//
//					if(k%10 == 0)
//						cout << k << endl;
//
//					int match = 0;
//
//					if(numEvents == 32)
//					{
//						for(int j = 0; j < 32; j++)
//						{
//							//fprintf(f2, "Para o evento %d foram aceites:\n", j);
//							for(int i = 0; i < numberSubs; i++)
//							{
//								unsigned int currSub = result[i];
//								//cout << currSub << endl;
//								if(ACTIVE_BIT(currSub, j)>0){
//									//fprintf(f2,"%d\n",i);
//									match++;
//								}
//							}
//						}
//						//cout << match << endl;
//						fprintf(f2,"Foram aceites %d subscricoes\n", match);
//					}
//					else
//					{
//						for(int i = 0; i < numberSubs; i++)
//						{
//							int currSub = result[i];
//
//							if(currSub > 0)
//								match++;
//						}
//						fprintf(f2,"Foram aceites %d subscricoes\n",match);
//						//cout << match << endl;
//					}
//					free(result);
//				}
//				cout << totaltime << endl;
//				f << totaltime << endl;
//				free(bitmap_CPU);
//	}
//				
//	/*free(str);
//	free(str2);*/
//	f.close();
//	fclose(f2);
//}
//
//
//
//
//void firstInit(){
//
//	//############ TAMANHO DA EXPERIENCIA ##################
//	numberClusters = 7;//indicar o numero de clusters
//	//numberPredicates = 1000;//numero de predicados
//	numberSubs = 500000;//numero de subscricoes
//	numIter = 100;//numero de itera��es
//
//	//#### PERCENTAGEM DE DIVISAO DOS PREDICADOS PELAS SUBSCRICOES #####
//	percPerC = 0.5;
//}
//
//
//void clearMemory()
//{
//	for(int i = 0; i <= numberClusters; i++)
//		free(clusters[i]);
//
//	free(clusters);
//}
//
//
//int
//main(int argc, char** argv)
//{
//	printf("Welcome to a matching publish/subscribe system! Made in CUDA!\n");
//
//	unsigned int timer;
//	cutilCheckError( cutCreateTimer(&timer));
//
//	cutStartTimer(timer);
//
//	usingshared = 0;
//	numberPredicates = 1000;
//	firstInit();
//	initSubs(0);
//	generateBitmapVar(1);
//	generateBitmapVar(32);
//	clearMemory();
//
//	usingshared = 0;
//	numberPredicates = 10000;
//	firstInit();
//	initSubs(0);
//	generateBitmapVar(1);
//	generateBitmapVar(32);
//	clearMemory();
//
//	usingshared = 0;
//	numberPredicates = 30000;
//	firstInit();
//	initSubs(0);
//	generateBitmapVar(1);
//	generateBitmapVar(32);
//	clearMemory();
//
//	cutStopTimer(timer);
//	printf("total time of execution = %f \n",cutGetTimerValue(timer)/(double)CLOCKS_PER_SEC);
//	cudaThreadExit();
//	
//	//cutilExit(argc, argv);
//	exit(EXIT_SUCCESS);
//
//	return 0;
//}
//
///*
////for(int i = 1; i < 100; i*=10)
//	//{
//		usingshared = 0;
//
//		numberPredicates = 10000;
//
//		firstInit();
//
//		initSubs(0);
//
//		mulInit = 100;
//		mulFinal = 100;
//
//		obtainMatchingSubsCPU(32,0);
//
//		//obtainMatchingSubsCPU(32,1);
//
//		clearMemory();
//	//}
//
//
//*/